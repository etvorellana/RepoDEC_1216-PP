/*
 * exampleCUDA-001.cu
 *
 * Copyright 2017 Esbel Tomas Valero Orellana <evalero@ninjapad>
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston,
 * MA 02110-1301, USA.
 *
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

#define SIZE 8192
#define BSIZE 16

__global__ void k_dgemm(double alpha, double* A, double* B, double beta, double* C, int msize, int TILES);

__host__ void dgemmCUDA( const char TA, const char TB,
			const int M, const int N, const int K,
			const double a, double *A, const int lda,
			double *B, const int ldb, const double b,
			double *C, const int ldc
			);

int main(int argc, char **argv)
{
	int i,j,k;

	int matSize = 32;
	double start, stop, dt, Dt;
	double gflop;

	int M , K, N;
	char ta, tb;
	const double alpha = 2.0; 
    const double beta = 2.0;
	int lda, ldb, ldc;

	FILE *desemp;

	if(argc > 1)
		desemp = fopen(argv[1], "w");
	else
		desemp = fopen("./desempenho.dat", "w");

	ta = tb = 'N';
	
	while (matSize <= SIZE){

		double *A = (double*) malloc( matSize * matSize * sizeof(double) );
		double *B = (double*) malloc( matSize * matSize * sizeof(double) );
		double *C = (double*) malloc( matSize * matSize * sizeof(double) );
		Dt = 0.0;

		M = K = N = matSize;
		lda = ldb = ldc = matSize;
		for(k=0; k<3; k++){
			for (i = 0; i < matSize; i++) {
				for (j = 0; j < matSize; j++) {
					//A[i*matSize+j] = (double)(rand()%3 - 1);
					A[i*matSize+j] = -1.0;
					//B[i*matSize+j] = (double)(rand()%9 - 4);
					B[i*matSize+j] = 0.0;
					//C[i*matSize+j] = ((double)rand()) / RAND_MAX;
					C[i*matSize+j] = 1.0;
				}
                B[i*matSize+i] = 1.0; // B é a matriz identidade
			}

			start = omp_get_wtime();
			dgemmCUDA(ta, tb, M, N, K, alpha, A, lda, B, ldb, beta, C, ldc);
			stop = omp_get_wtime();

			dt = stop - start;
			if(!k)
				Dt = dt;
			else if(Dt > dt)
				Dt = dt;
		}

		gflop = 2.0*matSize*matSize*matSize*0.000000001;
		printf("%f \n", C[0*ldc+0]);
		printf("%f \n", C[(matSize-1)*ldc+matSize-1]);
		printf("Result %d Ok!\n time: best - %f \n Size in GFLOP: %f\n Perfrmance: %f GFLOPS\n", matSize, Dt, gflop, gflop/Dt);
		fprintf(desemp, "%d\t%.12lf\t%.12lf\t%.12lf\n", matSize, Dt, gflop, gflop/Dt);
		fflush(stdout);
		fflush(desemp);
		free(A);
		free(B);
		free(C);
		matSize += 32;
	}
	fclose(desemp);
	return 0;
}



void dgemmCUDA( const char TA, const char TB,
			const int M, const int N, const int K,
			const double a, double *A, const int lda,
			double *B, const int ldb, const double b,
			double *C, const int ldc
			)
{
	int tSize = M * N * sizeof(double);
	double *Ad, *Bd, *Cd;

	// Alocar memória para as matrizes no device
	if ( hipSuccess != hipMalloc( (void**)&Ad, tSize ))
		printf( "Erro alocando A!\n" );
	if ( hipSuccess != hipMalloc( (void**)&Bd, tSize ))
		printf( "Erro alocando B!\n" );
	if ( hipSuccess != hipMalloc( (void**)&Cd, tSize ))
		printf( "Erro alocando C!\n" );
	// Copiar as matrizes para os respectivos espaços de memória no device
	hipError_t cudaErro;
	cudaErro = hipMemcpy(Ad, A, tSize, hipMemcpyHostToDevice);
	if ( hipSuccess != cudaErro)
		printf( "Erro copiando A!\n" );
	cudaErro = hipMemcpy(Bd, B, tSize, hipMemcpyHostToDevice);
	if ( hipSuccess != cudaErro)
		printf( "Erro copiando B!\n" );
	cudaErro = hipMemcpy(Cd, C, tSize, hipMemcpyHostToDevice);
	if ( hipSuccess != cudaErro)
		printf( "Erro copiando C!\n" );

	int TILES = M/BSIZE;

    dim3 dimGrade(TILES, TILES);
    dim3 dimBloco(BSIZE, BSIZE);

    // Chamada ao kernel que implementa a GEMM
    k_dgemm<<<dimGrade, dimBloco>>>(a, Ad, Bd, b, Cd, M, TILES);


	// Copiar a matriz resultante do divece para o host
	cudaErro = hipMemcpy(C, Cd, tSize, hipMemcpyDeviceToHost);
	if ( hipSuccess != cudaErro)
		printf( "Erro recuperando C!\n" );
	// Liberar a memória no device
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);

	return;
}


__global__ void k_dgemm(double alpha, double* A, double* B, double beta, double* C, int msize, int TILES)
{
    __shared__ double Ads[BSIZE][BSIZE];
    __shared__ double Bds[BSIZE][BSIZE];

    int tj = threadIdx.x;
    int ti = threadIdx.y;
    int bj = blockIdx.x;
    int bi = blockIdx.y;

    int i = bi*BSIZE + ti;
    int j = bj*BSIZE + tj;
    int m, n, k;

    double cValue = 0.0;

    for (m=0, n = 0; m < TILES; m++, n += BSIZE){
        Ads[ti][tj] = A [i*msize + n + tj];
        Bds[ti][tj] = B [(n+ti)*msize + j];
        __syncthreads();
        for (k = 0; k < BSIZE; k++)
            cValue += Ads[ti][k] * Bds[k][tj];
        __syncthreads();
    }

    C[i*msize + j] = alpha * cValue + beta * C[i*msize + j];
}